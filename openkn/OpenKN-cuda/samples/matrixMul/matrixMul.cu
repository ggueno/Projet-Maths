/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication and is exactly the same as
 * Chapter 7 of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <matrixMul_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);


void initMatrixMul(CudaParams * c) {
    CUT_DEVICE_INIT(0,NULL);

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    c->size_A = WA * HA;
    c->mem_size_A = sizeof(float) * c->size_A;
    c->h_A = (float*) malloc(c->mem_size_A);
    c->size_B = WB * HB;
    c->mem_size_B = sizeof(float) * c->size_B;
    c->h_B = (float*) malloc(c->mem_size_B);

    // initialize host memory
    randomInit(c->h_A, c->size_A);
    randomInit(c->h_B, c->size_B);

    // allocate device memory
    CUDA_SAFE_CALL(hipMalloc((void**) &(c->d_A), c->mem_size_A));
    CUDA_SAFE_CALL(hipMalloc((void**) &(c->d_B), c->mem_size_B));

    // copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(c->d_A, c->h_A, c->mem_size_A,
                              hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL(hipMemcpy(c->d_B, c->h_B, c->mem_size_B,
                              hipMemcpyHostToDevice) );

    // allocate device memory for result
    c->size_C = WC * HC;
    c->mem_size_C = sizeof(float) * c->size_C;
    CUDA_SAFE_CALL(hipMalloc((void**) &(c->d_C), c->mem_size_C));

    // allocate host memory for the result
    c->h_C = (float*) malloc(c->mem_size_C);
}


void runMatrixMul(CudaParams *c) {
    // create and start timer
    unsigned int timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // execute the kernel
    matrixMul<<< grid, threads >>>(c->d_C, c->d_A, c->d_B, WA, WB);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL(hipMemcpy(c->h_C, c->d_C, c->mem_size_C,
                              hipMemcpyDeviceToHost) );

    // stop and destroy timer
    CUT_SAFE_CALL(cutStopTimer(timer));
    printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    CUT_SAFE_CALL(cutDeleteTimer(timer));

    // compute reference solution
    float* reference = (float*) malloc(c->mem_size_C);
    computeGold(reference, c->h_A, c->h_B, HA, WA, WB);

    // check result
    CUTBoolean res = cutCompareL2fe(reference, c->h_C, c->size_C, 1e-6f);
    printf("Test %s \n", (1 == res) ? "PASSED" : "FAILED");
    if (res!=1) printDiff(reference, c->h_C, WC, HC);
}

void releaseMatrixMul(CudaParams * c) {
    // clean up memory
    free(c->h_A);
    free(c->h_B);
    free(c->h_C);
    //free(reference);
    CUDA_SAFE_CALL(hipFree(c->d_A));
    CUDA_SAFE_CALL(hipFree(c->d_B));
    CUDA_SAFE_CALL(hipFree(c->d_C));
}



// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}
